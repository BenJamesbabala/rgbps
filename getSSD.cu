#include "hip/hip_runtime.h"
/*
  Fast CUDA version that implements the following method. Use caller
  space variable pointers in an un-official way to prevent memory
  duplication on the GPU.

  Expects the following local variables to be present in the workspace
  of the calling function: psz,nValid,nxF,nyF,l2,c,imR,imG,imB,sc_rh
  
    nlF = bsxfun(@rdivide,c,sqrt(nxF.^2 + nyF.^2 + 1));
    nxF = nxF .* nlF; nyF = nyF .* nlF;

    sc_rh = sum( (l2(1,1)*nxF + l2(2,1)*nyF + l2(3,1)*nlF - imR).^2, 2);
    sc_rh = sc_rh + ...
	    sum( (l2(1,2)*nxF + l2(2,2)*nyF + l2(3,2)*nlF - imG).^2, 2);
    sc_rh = sc_rh + ...
	    sum( (l2(1,3)*nxF + l2(2,3)*nyF + l2(3,3)*nlF - imB).^2, 2);

  If any of the above variables don't exist, or aren't on the gpu when
  they're expected to be, this function might crash very badly!
	    
  Copyright (C) 2016, Ayan Chakrabarti <ayanc@ttic.edu>
*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <stdint.h>

#define F float

#define NUMT 1024

void __global__ getSSD(F * l2, F * nxf, F * nyf, F * cf,
		       F * imr, F * img, F * imb,
		       F * sc_rh,
		       int psz, int nV) {


	__shared__ F ell[9];
	F nxv, nyv, nlv, cv, val, sum;
	int i, j, nloc;
	

	i = threadIdx.x;

	if(i < 9) ell[i] = l2[i];
	__syncthreads();

	i += blockDim.x * blockIdx.x;
	if(i < nV) {
		cv = cf[i]; sum = 0; nloc = i;
		for(j = 0; j < psz; j++) {
			nxv = nxf[nloc]; nyv = nyf[nloc]; 
			nlv = cv / sqrtf(1.0+nxv*nxv+nyv*nyv); 
			nxv = nxv*nlv; nyv = nyv*nlv;

			val = ell[0]*nxv + ell[1]*nyv + ell[2]*nlv - imr[nloc];
			sum += val*val;

			val = ell[3]*nxv + ell[4]*nyv + ell[5]*nlv - img[nloc];
			sum += val*val;

			val = ell[6]*nxv + ell[7]*nyv + ell[8]*nlv - imb[nloc];
			sum += val*val;

			nloc += nV;
		}
		sc_rh[i] = sum;
	}

}


F * getGPUmem(const char * name) {

	const mxGPUArray * tmp;
	F * dptr;

	if(!mxIsGPUArray(mexGetVariablePtr("caller",name)))
		mexPrintf("%s is not on gpu!\n",name);

	tmp = mxGPUCreateFromMxArray(mexGetVariablePtr("caller",name));
	dptr = (F*) mxGPUGetDataReadOnly(tmp);
	mxGPUDestroyGPUArray(tmp);

	return (F*) dptr;
}

/* function getSSD */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	F * l2, * nxf, * nyf, * cf, * imr, * img, * imb, * sc_rh;
	int psz, nV, nB;

	psz = mxGetScalar(mexGetVariablePtr("caller","psz")); psz *= psz;
	nV = mxGetScalar(mexGetVariablePtr("caller","nValid"));

	nB = (nV+NUMT-1)/NUMT;


	l2 = getGPUmem("l2"); sc_rh = getGPUmem("sc_rh");
	nxf = getGPUmem("nxF");nyf = getGPUmem("nyF");cf = getGPUmem("c");
	imr = getGPUmem("imR");img = getGPUmem("imG");imb = getGPUmem("imB");

	getSSD<<<nB,NUMT>>>(l2,nxf,nyf,cf,imr,img,imb,sc_rh,psz,nV);
}
